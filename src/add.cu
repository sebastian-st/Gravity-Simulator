#include "hip/hip_runtime.h"
#include <stdio.h>

/**
 * GPU kernel function: Compute one acceleration component at an index derived from current threadIdx and blockIdx on the GPU
 */
__global__ void gpu_insert(double *x0, double *y0, double *masses, double *ax, double *ay, int start_idx, int end_idx, int N)
{
	// Define gravitational constant (arb. unit, example value!)
	double G = 1e-3;

	// Define softening scale of Plummer profile (arb. unit, example value!)
	double epsilon_squared = pow(0.1, 2);

	// Get current index for which the acceleration is to be computed
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < N)
	{
		// Get position
		double x_i = x0[i];
		double y_i = y0[i];

		double ax_sum = 0.;
		double ay_sum = 0.;

		// Directly sum the acceleration contributions from all particles without Fourier methods
		for (int j = start_idx; j < end_idx; ++j)
		{
			if (j == i)
				continue;

			// Get j-th particle position and mass
			double x_j = x0[j];
			double y_j = y0[j];
			double m_j = masses[j];

			// Intermediate quantities needed
			double dx = x_i - x_j;
			double dy = y_i - y_j;
			double dxsq = dx*dx;
			double dysq = dy*dy;
			double dr2 = dxsq + dysq + epsilon_squared;
			double dr = sqrt(dr2);
			double dr3 = dr*dr2;

			// Compute acceleration for the Plummer potential
			ax_sum += m_j * dx/dr3; 
			ay_sum += m_j * dy/dr3;
		}

		// Add contribution to sum
		ax[i] += -G*ax_sum;
		ay[i] += -G*ay_sum;

	}
}

/**
 * Auxiliary function
 */
int int_division_up(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

/**
 * Run the GPU computation, i.e. transfer data to GPU, call kernel, transfer data back
 */
void get_acceleration_vectors_GPU(double *x0, double *y0, double *masses, double *ax, double *ay, int N)
{
	// Define "device" quantities for coordinates, masses and accelerations
	double *dev_x0, *dev_y0, *dev_masses, *dev_ax, *dev_ay;
	int size = N *sizeof( double);
	hipMalloc((void**)&dev_x0, size);
	hipMalloc((void**)&dev_y0, size);
	hipMalloc((void**)&dev_masses, size);
	hipMalloc((void**)&dev_ax, size);
	hipMalloc((void**)&dev_ay, size);

	// Transfer data from host (CPU) to device (GPU)
	hipMemcpy(dev_x0, x0, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_y0, y0, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_masses, masses, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_ax, ax, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_ay, ay, size, hipMemcpyHostToDevice);

	// Set-up division for blocks and threads
	dim3 threads(N_t);
	dim3 blocks(int_division_up(N, N_t));

	// Run the kernel
	gpu_insert<<<blocks, threads>>>(dev_x0, dev_y0, dev_masses, dev_ax, dev_ay, 0, N, N);

	// Check for error
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	hipDeviceSynchronize();

	// Transfer data back and free the GPU memory
	hipMemcpy(ax, dev_ax, size, hipMemcpyDeviceToHost);
	hipMemcpy(ay, dev_ay, size, hipMemcpyDeviceToHost);
	hipFree(dev_ax);
	hipFree(dev_ay);
	hipFree(dev_x0);
	hipFree(dev_y0);
}
